#include "hip/hip_runtime.h"
#include "sphere.cuh"
#include "ray.cuh"

bool Sphere::intersect(const Ray& ray, Fragment& fragment/*, std::shared_ptr<Material> material*/) const {
	glm::vec3 oc = ray.origin() - m_center;
	auto a = glm::dot(ray.direction(), ray.direction());
	auto b = 2.f * glm::dot(oc, ray.direction());
	auto c = glm::dot(oc, oc) - m_radius * m_radius;
	auto d = b * b - 4.f * a * c;
	if (d <= 0.f)
		return false;

	float t = (-b - glm::sqrt(d)) / (2.f * a);
	if (t < ray.tmin() || t > ray.tmax()) {
		t = (-b + glm::sqrt(d)) / (2.f * a);
		if (t < ray.tmin() || t > ray.tmax())
			return false;
	}

	ray.tmax() = t;
	glm::vec3 pos = ray();
	glm::vec3 normal = glm::normalize(pos - m_center);
	if (glm::dot(normal, glm::normalize(-ray.direction())) < 0.f)
		normal = -normal;

	fragment = Fragment{ pos, normal, glm::normalize(-ray.direction()), m_radiance, m_emission, m_reflective };

	return true;
}