#include "hip/hip_runtime.h"
#include <iostream>

#include "utils.cuh"

#include "camera.cuh"
#include "ray.cuh"
#include "scene.cuh"
#include "sphere.cuh"
#include "triangle.cuh"


#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
	if (result) {
		std::cerr << "CUDA Error: " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "'" << std::endl;
		hipDeviceReset();
		exit(99);
	}
}

GPU_ONLY glm::vec3 filmic(glm::vec3 l) { return l / (l + 0.155f) * 1.019f; }

GPU_ONLY glm::vec3 transformSample(glm::vec3 n, glm::vec3 s) {
	glm::vec3 worldUp{ 0.f, 0.f, 1.f };
	if (glm::abs(n.z) >= 1.f - glm::epsilon<float>())
		worldUp = glm::vec3{ 1.f, 0.f, 0.f };
	glm::vec3 t = glm::normalize(glm::cross(worldUp, n));
	glm::vec3 b = glm::normalize(glm::cross(t, n));
	return glm::normalize(s.x * t + s.y * b + s.z * n);
}

GPU_ONLY glm::vec2 uniformDiskSample(hiprandState* local_rand_state) {
	float u = hiprand_uniform(local_rand_state) * 0.999f;
	float v = hiprand_uniform(local_rand_state) * 0.999f;
	float r = glm::sqrt(u);
	float theta = v * (2.f * glm::pi<float>());
	return r * glm::vec2{ glm::cos(theta), glm::sin(theta) };
}

GPU_ONLY glm::vec3 cosineHemisphereSample(glm::vec3 n, hiprandState* local_rand_state) {
	glm::vec2 xy = uniformDiskSample(local_rand_state);
	float z = glm::sqrt(glm::max(0.f, 1.f - xy.x * xy.x - xy.y * xy.y));
	return transformSample(n, glm::vec3{ xy.x, xy.y, z });
}

GPU_ONLY glm::vec3 uniformHemisphereSample(glm::vec3 n, hiprandState* local_rand_state) {
	float u = hiprand_uniform(local_rand_state) * 0.999f;
	float v = hiprand_uniform(local_rand_state) * 0.999f;
	float r = glm::sqrt(glm::max(0.f, 1.f - u * u));
	float phi = 2.f * glm::pi<float>() * v;
	
	glm::vec3 ls{ r * glm::cos(phi), r * glm::sin(phi), u };
	return transformSample(n, ls);
}

GPU_ONLY glm::vec3 li(const Ray& ray, Scene** scene, int bounces, hiprandState* local_rand_state) {
	Ray curRay = ray;
	glm::vec3 curRadiance{ 1.f };
	glm::vec3 finalColor{ 0.f };
	float curFactor = 1.f;
	float prr = 1.f;
	glm::vec3 throughput{ 1.f };
	while (true) {
		if (bounces-- <= 0)
			prr = glm::min(glm::max(throughput.r, glm::max(throughput.g, throughput.b)), 0.99f);

		if (hiprand_uniform(local_rand_state) * 0.999f >= prr)
			return finalColor;

		Fragment frag;
		if (!(*scene)->intersect(curRay, frag))
			return finalColor + (*scene)->skyColor(ray.direction()) * curRadiance;

		glm::vec3 wi{ 0.f };
		glm::vec3 radiance{ 1.f };
		if (frag.reflective())
			wi = glm::reflect(-frag.wo(), frag.normal());
		else {
			wi = cosineHemisphereSample(frag.normal(), local_rand_state);
			float cosTheta = glm::max(glm::dot(frag.normal(), wi), 0.f);
			float pdf = cosTheta / glm::pi<float>();
			radiance = cosTheta * (frag.radiance() / glm::pi<float>()) / pdf;
		}

		finalColor += curRadiance * frag.emission(); // to change before updating the curRadiance => I = R0 * E1 + R0 * R1 * E2...
		curRay = Ray{ frag.position() + 0.01f * frag.normal(), wi };
		curRadiance *= curFactor * radiance / prr;
		curFactor *= 0.5f;
	}
	return finalColor;
}

KERNEL void initializeScene(Scene** scene, Camera** camera, Shape** shapes, int nx, int ny) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		//light
		*(shapes + 0) = new Triangle{ glm::vec3{ -5.f, 19.9f, 5.f }, glm::vec3{ -5.f, 19.9f, -5.f }, glm::vec3{ 5.f, 19.9f, -5.f }, glm::vec3{ 1.f }, glm::vec3{ 15.f } };
		*(shapes + 1) = new Triangle{ glm::vec3{ -5.f, 19.9f, 5.f }, glm::vec3{ 5.f, 19.9f, -5.f }, glm::vec3{ 5.f, 19.9f, 5.f }, glm::vec3{ 1.f }, glm::vec3{ 15.f } };

		//roof
		*(shapes + 2) = new Triangle{ glm::vec3{ -20.f, 20.f, 20.f }, glm::vec3{ -20.f, 20.f, -20.f }, glm::vec3{ 20.f, 20.f, -20.f }, glm::vec3{ 0.8f } };
		*(shapes + 3) = new Triangle{ glm::vec3{ -20.f, 20.f, 20.f }, glm::vec3{ 20.f, 20.f, 20.f }, glm::vec3{ 20.f, 20.f, -20.f }, glm::vec3{ 0.8f } };

		//ground
		*(shapes + 4) = new Triangle{ glm::vec3{ -20.f, -20.f, 20.f }, glm::vec3{ -20.f, -20.f, -20.f }, glm::vec3{ 20.f, -20.f, -20.f }, glm::vec3{ 0.8f } };
		*(shapes + 5) = new Triangle{ glm::vec3{ -20.f, -20.f, 20.f }, glm::vec3{ 20.f, -20.f, 20.f }, glm::vec3{ 20.f, -20.f, -20.f }, glm::vec3{ 0.8f } };

		//back wall
		*(shapes + 6) = new Triangle{ glm::vec3{ -20.f, 20.f, 20.f }, glm::vec3{ 20.f, 20.f, 20.f }, glm::vec3{ 20.f, -20.f, 20.f }, glm::vec3{ 0.8f } };
		*(shapes + 7) = new Triangle{ glm::vec3{ -20.f, 20.f, 20.f }, glm::vec3{ 20.f, -20.f, 20.f }, glm::vec3{ -20.f, -20.f, 20.f }, glm::vec3{ 0.8f } };

		//red wall
		*(shapes + 8) = new Triangle{ glm::vec3{ -20.f, 20.f, -20.f }, glm::vec3{ -20.f, 20.f, 20.f }, glm::vec3{ -20.f, -20.f, 20.f }, glm::vec3{ 0.8f, 0.1f, 0.1f } };
		*(shapes + 9) = new Triangle{ glm::vec3{ -20.f, 20.f, -20.f }, glm::vec3{ -20.f, -20.f, 20.f }, glm::vec3{ -20.f, -20.f, -20.f }, glm::vec3{ 0.8f, 0.1f, 0.1f } };

		//green wall
		*(shapes + 10) = new Triangle{ glm::vec3{ 20.f, 20.f, -20.f }, glm::vec3{ 20.f, 20.f, 20.f }, glm::vec3{ 20.f, -20.f, 20.f }, glm::vec3{ 0.1f, 0.8f, 0.1f } };
		*(shapes + 11) = new Triangle{ glm::vec3{ 20.f, 20.f, -20.f }, glm::vec3{ 20.f, -20.f, 20.f }, glm::vec3{ 20.f, -20.f, -20.f }, glm::vec3{ 0.1f, 0.8f, 0.1f } };

		//spheres
		*(shapes + 12) = new Sphere{ glm::vec3{ 7.5f, -13.8f, -7.f }, 6.f, glm::vec3{ 1.f } };
		*(shapes + 13) = new Sphere{ glm::vec3{ -3.5f, -10.8f, 6.f }, 9.f, glm::vec3{ 0.8f }, glm::vec3{ 0.f }, true };

		*scene = new Scene{ shapes, 14 };
		*camera = new Camera{ nx, ny, glm::vec3{ 0.f, 0.f, -40.f } };
	}
}

KERNEL void freeScene(Scene** scene, Camera** camera, Shape** shapes) {
	for (int i = 0; i < 14; ++i)
		delete* (shapes + i);
	delete *scene;
	delete *camera;
}

KERNEL void render(glm::vec3* fb, int nx, int ny, int spp, int maxBounces, Scene** scene, Camera** camera, hiprandState* rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if (i >= nx || j >= ny) return;
	int px = j * nx + i;
	hiprandState local_rand_state = rand_state[px];
	hiprand_init(1984, px, 0, &local_rand_state);

	glm::vec3 color{ 0.f };
	for (int _ = 0; _ < spp; ++_) {
		auto u = static_cast<float>(i) + hiprand_uniform(&local_rand_state);
		auto v = static_cast<float>(j) + hiprand_uniform(&local_rand_state);
		Ray ray = (*camera)->castRay(u, v);
		color += li(ray, scene, maxBounces, &local_rand_state);
	}
	rand_state[px] = local_rand_state;
	color /= static_cast<float>(spp);
	fb[px] = glm::sqrt(glm::pow(filmic(color), glm::vec3{ 2.2f }));
}

int main() {
	int nx = 512, ny = 512;
	int spp = 1024;
	int maxBounces = 4;

	int numpx = nx * ny;

	glm::vec3* fb;
	Shape** dshapes;
	Scene** dscene;
	Camera** dcamera;
	hiprandState* drand_state;

	checkCudaErrors(hipMallocManaged((void**)&fb, numpx * sizeof(glm::vec3)));
	checkCudaErrors(hipMalloc((void**)&dshapes, 14 * sizeof(Shape *)));
	checkCudaErrors(hipMalloc((void**)&dscene, sizeof(Scene *)));
	checkCudaErrors(hipMalloc((void**)&dcamera, sizeof(Camera *)));
	checkCudaErrors(hipMalloc((void**)&drand_state, numpx * sizeof(hiprandState)));

	initializeScene<<<1, 1>>>(dscene, dcamera, dshapes, nx, ny);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	int tx = 8, ty = 8;
	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);
	render<<<blocks, threads>>>(fb, nx, ny, spp, maxBounces, dscene, dcamera, drand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	freeScene<<<1, 1>>>(dscene, dcamera, dshapes);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	std::cout << "Image rendered 100%. Writing to file." << std::endl;

	output(fb, nx, ny, "out.png");

	checkCudaErrors(hipFree(fb));
	checkCudaErrors(hipFree(dscene));
	checkCudaErrors(hipFree(dcamera));
	checkCudaErrors(hipFree(dshapes));
	checkCudaErrors(hipFree(drand_state));

	return 0;
}