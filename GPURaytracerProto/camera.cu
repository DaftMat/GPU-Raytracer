#include "hip/hip_runtime.h"
#include "camera.cuh"
#include "ray.cuh"

Camera::Camera(float imgWidth, float imgHeight, glm::vec3 position, glm::vec3 at, glm::vec3 worldUp, float fov) :
	m_aspect{ imgWidth / imgHeight },
	m_position{ position },
	m_fov{ fov },
	m_imgWidth{ imgWidth },
	m_imgHeight{ imgHeight }
{
	m_front = glm::normalize(at - position);
	m_right = glm::normalize(glm::cross(worldUp, m_front));
	m_up = glm::normalize(glm::cross(m_front, m_right));
	m_center = 1.f / tanf((m_fov * glm::pi<float>() / 180.f) * 0.5f) * m_front;

}

Ray Camera::castRay(float u, float v) const {
	float aspectInv = 1.f / m_aspect;

	float deltay = 1.f / (m_imgHeight * 0.5f);   //! one pixel size
	glm::vec3 dy = deltay * aspectInv * m_up; //! one pixel step
	glm::vec3 raydeltay = (0.5f - m_imgHeight * 0.5f) / (m_imgHeight * 0.5f) * aspectInv * m_up;

	float deltax = 1.f / (m_imgWidth * 0.5f);
	glm::vec3 dx = deltax * m_right;
	glm::vec3 raydeltax = (0.5f - m_imgWidth * 0.5f) / (m_imgWidth * 0.5f) * m_right;

	glm::vec3 corner = m_center + raydeltax - raydeltay;
	glm::vec3 rayDir = corner + u * dx - v * dy;

	return Ray{ m_position, glm::normalize(rayDir) };
}