#include "hip/hip_runtime.h"
#include "triangle.cuh"
#include "ray.cuh"

bool Triangle::intersect(const Ray& ray, Fragment& frag) const {
	glm::vec3 A = m_v0 - m_v2;
	glm::vec3 B = m_v1 - m_v2;
	glm::vec3 T = ray.origin() - m_v2;

	glm::vec3 normal = glm::normalize(glm::cross(B, A));
	if (glm::abs(glm::dot(normal, ray.direction())) < glm::epsilon<float>())
		return false;

	glm::vec3 p = glm::cross(ray.direction(), B);
	glm::vec3 q = glm::cross(T, A);

	float d = glm::dot(p, A);
	if (glm::abs(d) < glm::epsilon<float>())
		return false;
	float u = (1.f / d) * glm::dot(p, T);
	if (u < 0.f)
		return false;
	float v = (1.f / d) * glm::dot(q, ray.direction());
	if (v < 0.f || (u + v) > 1.f)
		return false;

	float t = (1.f / d) * glm::dot(q, B);
	if (t < ray.tmin() || t > ray.tmax())
		return false;

	ray.tmax() = t;
	glm::vec3 pos = ray();
	frag = Fragment{ pos, normal, glm::normalize(-ray.direction()), m_radiance, m_emission, m_reflective };

	return true;
}